#include "hip/hip_runtime.h"
#include <unistd.h>
#include <sys/time.h>
#include "svt_utils.h"
#include <math.h>

#include <sched.h>
#include "semaphore.c"
#include <thrust/device_vector.h>


// global variables
int VERBOSE = 0;
int TIMER = 0;

// CUDA timer macros
hipEvent_t c_start, c_stop;

inline void start_time() {
  if ( TIMER ) {
    hipEventCreate(&c_start);
    hipEventCreate(&c_stop);
    hipEventRecord(c_start, 0);
  }
}

inline float stop_time(const char *msg) {
  float elapsedTime = 0;
  if ( TIMER ) { 
    hipEventRecord(c_stop, 0);
    hipEventSynchronize(c_stop);
    hipEventElapsedTime(&elapsedTime, c_start, c_stop);
    if ( VERBOSE )
      printf("Time to %s: %.3f ms\n", msg, elapsedTime);
  }
  return elapsedTime;
}

// calculate mean and stdev on an array of count floats
void get_mean(float *times_array, int count, float *mean, float *stdev) {

  int j;
  float sum = 0;
  float sumsqr = 0;

  *mean = *stdev = 0;

  for (j=0; j < count; j++) {
    sum += times_array[j];
    sumsqr += pow(times_array[j],2);
  }

  *mean = sum/(float)count;

  *stdev = sqrt(abs((sumsqr/(float)count) - pow(*mean,2)));
}

__global__ void init_arrays_GPU (fout_arrays* fout_dev, evt_arrays* evt_dev, int* events ) {

  int ie, ir, ip;

  *events = 0;

  ie = blockIdx.x; // events index
  ir = blockIdx.y; // roads index
  ip = threadIdx.x; // NSVX_PLANE+1

  // initialize evt arrays....
  evt_dev->evt_nroads[ie] = 0;
  evt_dev->evt_ee_word[ie] = 0;
  evt_dev->evt_err_sum[ie] =0;

  evt_dev->evt_zid[ie][ir] = 0;
  evt_dev->evt_err[ie][ir] = 0;
  evt_dev->evt_cable_sect[ie][ir] = 0;
  evt_dev->evt_sect[ie][ir] = 0;
  evt_dev->evt_road[ie][ir] = 0;

  evt_dev->evt_nhits[ie][ir][ip] = 0;

  // initialize fout arrays....
  fout_dev->fout_ntrks[ie] = 0;
  fout_dev->fout_parity[ie] = 0;
  fout_dev->fout_ee_word[ie] = 0;
  fout_dev->fout_err_sum[ie] = 0;
  fout_dev->fout_cdferr[ie] = 0;
  fout_dev->fout_svterr[ie] = 0;
    
}


void setedata_GPU(tf_arrays_t tf, struct extra_data *edata_dev) {

  int len;
  len = SVTSIM_NBAR * FITBLOCK * sizeof(int);
  MY_CUDA_CHECK(hipMemcpy(edata_dev->whichFit, tf->whichFit, len, hipMemcpyHostToDevice));
  len = NFITPAR * (DIMSPA+1) * SVTSIM_NBAR * FITBLOCK * sizeof(long long int);
  MY_CUDA_CHECK(hipMemcpy(edata_dev->lfitparfcon, tf->lfitparfcon, len, hipMemcpyHostToDevice));
  len = NEVTS * sizeof(int);
  MY_CUDA_CHECK(hipMemcpy(edata_dev->wedge, tf->wedge, len, hipMemcpyHostToDevice));

}


int svt_GPU(tf_arrays_t tf, struct extra_data *edata_dev, unsigned int *data_in, int n_words, float *timer, int nothrust, unsigned int* dataout) {

  int tEvts=0;
  int ndata=0;
  dim3 blocks(NEVTS,MAXROAD);

  start_time();
  // Cuda Malloc
  int* d_tEvts;
  MY_CUDA_CHECK(hipMalloc((void**)&d_tEvts, sizeof(int)));
  int* ndata_dev;
  MY_CUDA_CHECK(hipMalloc((void**)&ndata_dev, sizeof(int)));
  struct evt_arrays* evt_dev;
  MY_CUDA_CHECK(hipMalloc((void**)&evt_dev, sizeof(evt_arrays)));
  struct fep_arrays *fep_dev;
  MY_CUDA_CHECK(hipMalloc((void**)&fep_dev, sizeof(fep_arrays)));
  struct fit_arrays *fit_dev;
  MY_CUDA_CHECK(hipMalloc((void**)&fit_dev, sizeof(fit_arrays)));
  struct fout_arrays *fout_dev;
  MY_CUDA_CHECK(hipMalloc((void**)&fout_dev, sizeof(fout_arrays)));
  unsigned int *dataout_dev;
  MY_CUDA_CHECK(hipMalloc((void**)&dataout_dev, n_words*sizeof(int)));

  // initialize structures
  init_arrays_GPU<<<blocks, NSVX_PLANE+1>>>(fout_dev, evt_dev, d_tEvts);

  if ( nothrust ) { // use pure cuda version of unpack
    
    unsigned int *d_data_in;
    long sizeW = sizeof(int) * n_words;
    hipMalloc((void **)&d_data_in, sizeW);

    hipMemcpy(d_data_in, data_in, sizeW, hipMemcpyHostToDevice);

    timer[0] = stop_time("input copy and initialize");

    start_time();

    gf_unpack_cuda_GPU(d_data_in, n_words, evt_dev, d_tEvts );

    hipFree(d_data_in);

  } else { // use thrust version of unpack

    thrust::device_vector<unsigned int> d_vec(n_words+1);
    d_vec[0] = 0;
    thrust::copy(data_in, data_in + n_words, d_vec.begin()+1);

    timer[0] = stop_time("input copy and initialize");

    start_time();

    gf_unpack_thrust_GPU(d_vec, n_words, evt_dev, d_tEvts );

  } 

  timer[1] = stop_time("input unpack");

  MY_CUDA_CHECK(hipMemcpy(&tEvts, d_tEvts, sizeof(int), hipMemcpyDeviceToHost));
  tf->totEvts = tEvts;
 
  // Fep comb and set
  start_time();  
  gf_fep_GPU( evt_dev, fep_dev, tEvts );
  timer[2] =stop_time("compute fep combinations");

  // Fit and set Fout
  start_time();
  gf_fit_GPU(fep_dev, evt_dev, edata_dev, fit_dev, fout_dev, tEvts, dataout_dev, ndata_dev);
  timer[3] = stop_time("fit data and set output");

  // Output copy DtoH
  start_time();

  MY_CUDA_CHECK(hipMemcpy(&ndata, ndata_dev, sizeof(int), hipMemcpyDeviceToHost));
  MY_CUDA_CHECK(hipMemcpy(dataout, dataout_dev, ndata * sizeof(int), hipMemcpyDeviceToHost));

  MY_CUDA_CHECK( hipFree(evt_dev) );
  MY_CUDA_CHECK( hipFree(fep_dev) );
  MY_CUDA_CHECK( hipFree(fit_dev) );
  MY_CUDA_CHECK( hipFree(fout_dev));
  MY_CUDA_CHECK( hipFree(d_tEvts));
  MY_CUDA_CHECK( hipFree(ndata_dev));
  MY_CUDA_CHECK( hipFree(dataout_dev));
  timer[4] = stop_time("copy output (DtoH)");

  return ndata;

}

void help(char* prog) {

  printf("Use %s [-i fileIn] [-o fileOut] [-s cpu || gpu] [-l #loops] [-u] [-v] [-t] [-p priority] [-h] \n\n", prog);
  printf("  -i fileIn       Input file (Default: hbout_w6_100evts).\n");
  printf("  -o fileOut      Output file (Default: gfout.txt).\n");
  printf("  -s cpu || gpu   Switch between CPU or GPU version (Default: gpu).\n");
  printf("  -l loops        Number of executions (Default: 1).\n");
  printf("  -u              Use pure cuda version for unpack (Default: use thrust version).\n");
  printf("  -v              Print verbose messages.\n");
  printf("  -t              Calculate timing.\n");
  printf("  -p priority     Set scheduling priority to <priority> and cpu affinity - you nedd to be ROOT - (Default: disable).\n");
  printf("  -h              This help.\n");

}

int main(int argc, char* argv[]) {

  int c;
  char* fileIn = "hbout_w6_100evts";
  char* fileOut = "gfout.txt";
  char* where = "gpu";
  int N_LOOPS = 1;
  int PRIORITY = 0;
  int NOTHRUST = 0;

  while ( (c = getopt(argc, argv, "i:s:o:l:uvtp:h")) != -1 ) {
    switch(c) {
      case 'i': 
        fileIn = optarg;
	      break;
      case 'o':
        fileOut = optarg;
        break;
	    case 's': 
        where = optarg;
	      break;
      case 'l':
        N_LOOPS = atoi(optarg);
        break;
      case 'v':
        VERBOSE = 1;
        break;
      case 'u':
        NOTHRUST = 1;
        break;
      case 't':
        TIMER = 1;
        break;
      case 'p':
        PRIORITY = atoi(optarg);
        break;
      case 'h':
        help(argv[0]);
        return 0;
    }
  }

  if (access(fileIn, 0) == -1) {
    printf("ERROR: File %s doesn't exist.\n", fileIn);
    return 1;
  }

  int semid;
  if ( PRIORITY ) {
    // lock control so no one else can run at the same time and crash the machine
    key_t key = (key_t) 0xdeadface;

    if ((semid = initsem(key, 1)) == -1) {
        perror("initsem");
        exit(1);
    }
    printf("Trying to gain control...\n");
    lock(semid);

    // set scheduling priority & CPU affinity
    struct sched_param p;
    p.sched_priority = PRIORITY;
    if (sched_setscheduler(0, SCHED_FIFO, &p)) {
      perror("setscheduler");
      return -1;
    }
    if (sched_getparam(0, &p) == 0)
      printf("Running with scheduling priority = %d\n", p.sched_priority);

    unsigned long mask;
    if (sched_getaffinity(0, sizeof(mask), (cpu_set_t*)&mask) < 0) {
      perror("sched_getaffinity");
    }
    printf("my affinity mask is: %08lx\n", mask);

    mask = 1; // processor 1 only
    if (sched_setaffinity(0, sizeof(mask), (cpu_set_t*)&mask) < 0) {
      perror("sched_setaffinity");
      return -1;
    }

    if (sched_getaffinity(0, sizeof(mask), (cpu_set_t*)&mask) < 0) {
      perror("sched_getaffinity");
    }
    printf("my affinity mask is: %08lx\n", mask);
  }

  // Do we want to skip the first "skip" runs from mean calculation?
  int skip = 0;
  int n_iters = N_LOOPS+skip;

  float initg = 0;
  float fcon = 0;
  float timerange = 0;
  float ptime[5];
  float ptime_cpu[3];
  float times_array[6][N_LOOPS];
  float times_array_cpu[4][N_LOOPS];

  struct timeval time_start, time_stop;
  struct timeval tBegin, tEnd;
  struct timeval ptBegin, ptEnd;


  if ( strcmp(where,"gpu") == 0 ) { // GPU

    if ( TIMER ) gettimeofday(&tBegin, NULL);
    
    // this is just to measure time to initialize GPU
    hipEvent_t     init;
    MY_CUDA_CHECK( hipEventCreate( &init ) );
    if ( TIMER ) {
      gettimeofday(&tEnd, NULL);
      initg = ((tEnd.tv_usec + 1000000 * tEnd.tv_sec) - (tBegin.tv_usec + 1000000 * tBegin.tv_sec))/1000000.0;  
    }
  }

  
  // read input file
  FILE* hbout = fopen(fileIn,"r");

  if ( hbout == NULL ) {
    printf("ERROR: Cannot open input file\n");
    exit(1);
  }

  unsigned int hexaval;
  unsigned int *data_send = (unsigned int*)malloc(2500000*sizeof(unsigned));
  if ( data_send == (unsigned int*) NULL ) {
    perror("malloc");
    return 2;
  }
  
  char word[16];
  int k=0; // number of words read
  if ( VERBOSE ) printf("Reading input file %s... ", fileIn);
  while (fscanf(hbout, "%s", word) != EOF) {
    hexaval = strtol(word,NULL,16);
    data_send[k] = hexaval;
    k++;
  }

  fclose(hbout);

  int outword;
  unsigned int *dataout = (unsigned int*)malloc(k*sizeof(unsigned));

  tf_arrays_t tf;
  gf_init(&tf);
  svtsim_fconread(tf);

  if ( TIMER ) gettimeofday(&tBegin, NULL);
  
  struct extra_data *edata_dev;

  if ( strcmp(where,"cpu") != 0 ) { // GPU
    if ( TIMER ) start_time();
    MY_CUDA_CHECK(hipMalloc((void**)&edata_dev, sizeof(struct extra_data)));
    setedata_GPU(tf, edata_dev);
    if ( TIMER ) fcon = stop_time("Copy detector configuration data");
  }

  while (n_iters--) {

    if ( strcmp(where,"cpu") == 0 ) { // CPU
      if ( TIMER ) gettimeofday(&time_start, NULL);
      
      if ( VERBOSE ) printf("Start working on CPU..... \n");
      
      if ( TIMER ) gettimeofday(&ptBegin, NULL);

      gf_fep_unpack(tf, k, data_send);

      if ( TIMER) {
        gettimeofday(&ptEnd, NULL);
        timerange = ((ptEnd.tv_usec + 1000000 * ptEnd.tv_sec) - (ptBegin.tv_usec + 1000000 * ptBegin.tv_sec))/1000.0;
        if ( VERBOSE ) printf("Time to CPU unpack: %.3f ms\n", timerange);
        ptime_cpu[0] = timerange;
 
        gettimeofday(&ptBegin, NULL);
      }
        
      gf_fep_comb(tf);
      
      if ( TIMER) {
        gettimeofday(&ptEnd, NULL);
        timerange = ((ptEnd.tv_usec + 1000000 * ptEnd.tv_sec) - (ptBegin.tv_usec + 1000000 * ptBegin.tv_sec))/1000.0;
        if ( VERBOSE )  printf("Time to CPU comb: %.3f ms\n", timerange);
        ptime_cpu[1] = timerange;

        gettimeofday(&ptBegin, NULL);
      }

      gf_fit(tf);
      gf_comparator(tf);
      
      if ( TIMER) {
        gettimeofday(&ptEnd, NULL);
        timerange = ((ptEnd.tv_usec + 1000000 * ptEnd.tv_sec) - (ptBegin.tv_usec + 1000000 * ptBegin.tv_sec))/1000.0;
        if ( VERBOSE ) printf("Time to CPU fit: %.3f ms\n", timerange);
        ptime_cpu[2] = timerange;
        gettimeofday(&time_stop, NULL); 
      }
      if ( VERBOSE ) printf(".... fits %d events! \n", tf->totEvts);
      

    } else { // GPU
      if ( VERBOSE ) printf("Start working on GPU...\n");
      if ( TIMER ) gettimeofday(&time_start, NULL);  
      outword = svt_GPU(tf, edata_dev, data_send, k, ptime, NOTHRUST, dataout);
      if ( TIMER ) gettimeofday(&time_stop, NULL);
    }

    if ( TIMER ) {
      if ( n_iters < N_LOOPS ) { // skip the first "skip" iterations
        timerange = ((time_stop.tv_usec + 1000000 * time_stop.tv_sec) - (time_start.tv_usec + 1000000 * time_start.tv_sec))/1000.0;
        if ( strcmp(where,"cpu") != 0 ) { // GPU
          times_array[0][n_iters] = timerange;
          for (int t=1; t < 6; ++t) 
            times_array[t][n_iters] = ptime[t-1];
        } else { //CPU
          times_array_cpu[0][n_iters] = timerange; 
          for (int t=1; t < 4; ++t)
            times_array_cpu[t][n_iters] = ptime_cpu[t-1];
        }
      }
    }
  } // end iterations

  if ( strcmp(where,"cpu") != 0 ) {
    MY_CUDA_CHECK(hipFree(edata_dev));
  }

  if ( TIMER ) {
    gettimeofday(&tEnd, NULL);
    timerange = ((tEnd.tv_usec + 1000000 * tEnd.tv_sec) - (tBegin.tv_usec + 1000000 * tBegin.tv_sec))/1000.0;
    if ( VERBOSE ) printf("Time to complete all: %.3f ms\n", timerange); 
  }

  // write output file
  FILE* OUTCHECK = fopen(fileOut, "w");
  if ( strcmp(where,"cpu") == 0 )  // CPU
    for (int i=0; i < tf->out->ndata; i++)
      fprintf(OUTCHECK,"%.6x\n", tf->out->data[i]);
  else // GPU
    for (int i=0; i < outword; i++)
      fprintf(OUTCHECK,"%.6x\n", dataout[i]);
  fclose(OUTCHECK);
  
  // write file with times
  if ( TIMER ) {
    char fileTimes[1024];
    FILE *ft;
    if ( strcmp(where,"cpu") != 0 ) { // GPU
      float mean[6];
      float stdev[6];
      for (int t=0; t < 6; ++t) 
        get_mean(times_array[t], N_LOOPS, &mean[t], &stdev[t]);

      sprintf(fileTimes, "ListTimesGPU-Evts_%d_Loops_%d.txt", NEVTS, N_LOOPS);

      ft = fopen(fileTimes, "w");
      fprintf(ft,"# #NEvts: %d, Loops: %d, mean: %.3f ms, stdev: %.3f ms\n", NEVTS, N_LOOPS, mean[0], stdev[0]);
      fprintf(ft,"# initialize GPU: %.3f ms; copy detector configuration data: %.3f ms\n", initg, fcon);
      fprintf(ft,"# input copy and initialize        --> mean: %.3f ms, stdev: %.3f ms\n", mean[1], stdev[1]);
      fprintf(ft,"# input unpack                     --> mean: %.3f ms, stdev: %.3f ms\n", mean[2], stdev[2]);
      fprintf(ft,"# compute fep combinations         --> mean: %.3f ms, stdev: %.3f ms\n", mean[3], stdev[3]);
      fprintf(ft,"# fit data and set output          --> mean: %.3f ms, stdev: %.3f ms\n", mean[4], stdev[4]);
      fprintf(ft,"# copy output (DtoH)               --> mean: %.3f ms, stdev: %.3f ms\n", mean[5], stdev[5]);
    

      for (int j=0 ; j < (N_LOOPS); j++) {
        for (int t=0; t < 6; ++t)
          fprintf(ft,"%.3f ",times_array[t][j]);
        fprintf(ft,"\n");
      }
    } else { // CPU
      float mean[4];
      float stdev[4];
      for (int t=0; t < 4; ++t)
        get_mean(times_array_cpu[t], N_LOOPS, &mean[t], &stdev[t]);

      sprintf(fileTimes, "ListTimesCPU-Evts_%d_Loops_%d.txt", NEVTS, N_LOOPS);

      ft = fopen(fileTimes, "w");
      fprintf(ft,"# #NEvts: %d, Loops: %d, mean: %.3f ms, stdev: %.3f ms\n", NEVTS, N_LOOPS, mean[0], stdev[0]);
      fprintf(ft,"# input unpack                     --> mean: %.3f ms, stdev: %.3f ms\n", mean[1], stdev[1]);
      fprintf(ft,"# compute fep combinations         --> mean: %.3f ms, stdev: %.3f ms\n", mean[2], stdev[2]);
      fprintf(ft,"# fit data and set output          --> mean: %.3f ms, stdev: %.3f ms\n", mean[3], stdev[3]);

      for (int j=0 ; j < (N_LOOPS); j++) {
        for (int t=0; t < 4; ++t)
          fprintf(ft,"%.3f ",times_array_cpu[t][j]);
        fprintf(ft,"\n");
      }
    }

    fclose(ft);

    printf("All done. See %s for timing.\n", fileTimes);
  }


  if ( PRIORITY ) {
    if ( VERBOSE ) printf("Unlocking control...\n");
    unlock(semid);
  }

  free(data_send);
  free(tf);

  return 0;
}
